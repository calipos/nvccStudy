#include <iostream>


#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

// cuda
#include <hip/hip_runtime.h>
#include <hipblas.h>

///usr/local/cuda-8.0/bin/nvcc other.cu -std=c++11    -I/usr/local/cuda-8.0/include    -I/media/hdd/lbl_trainData/softwarePackage/cudnn-8.0-linux-x64-v7/include  -L/usr/local/cuda-8.0/lib64  -lcublas -lcudart -L/media/hdd/lbl_trainData/softwarePackage/cudnn-8.0-linux-x64-v7/lib64  -lcudnn  -gencode arch=compute_61,code=sm_61  


#define N       4
#define Value   2
#define checkCudaAPIErrors(F) if ((F) != hipSuccess) \
{ printf("Error at line %d in file %s: %s\n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError())); exit(-1); }

void initArray(char * a, const int size) {
    for (int i = 0; i < size; ++i) 
	{
		a[i] = Value;
    }
}
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
		case HIPBLAS_STATUS_SUCCESS:
			return "CUBLAS_STATUS_SUCCESS";

		case HIPBLAS_STATUS_NOT_INITIALIZED:
			return "CUBLAS_STATUS_NOT_INITIALIZED";

		case HIPBLAS_STATUS_ALLOC_FAILED:
			return "CUBLAS_STATUS_ALLOC_FAILED";

		case HIPBLAS_STATUS_INVALID_VALUE:
			return "CUBLAS_STATUS_INVALID_VALUE";

		case HIPBLAS_STATUS_ARCH_MISMATCH:
			return "CUBLAS_STATUS_ARCH_MISMATCH";

		case HIPBLAS_STATUS_MAPPING_ERROR:
			return "CUBLAS_STATUS_MAPPING_ERROR";

		case HIPBLAS_STATUS_EXECUTION_FAILED:
			return "CUBLAS_STATUS_EXECUTION_FAILED";

		case HIPBLAS_STATUS_INTERNAL_ERROR:
			return "CUBLAS_STATUS_INTERNAL_ERROR";

		case HIPBLAS_STATUS_NOT_SUPPORTED:
			return "CUBLAS_STATUS_NOT_SUPPORTED";

		case HIPBLAS_STATUS_UNKNOWN:
			return "CUBLAS_STATUS_LICENSE_ERROR";
	}

	return "<unknown>";
}

#define checkcuBlasError(F) if ((F) != HIPBLAS_STATUS_SUCCESS) \
{ printf("Error at line %d in file %s: %s\n", __LINE__, __FILE__, _cudaGetErrorEnum(F)); exit(-1); }

/** @main function ****************
**********************************/
int main(int argc, char** argv)
{
    // test_count
    int iters = 1;

    int alpha = 1;
    int beta  = 0;

    float TFlops;
	hipblasStatus_t cublasStat;

    int n[N] = {8,  512,  512,  512};
    int k[N] = {12, 2048, 2048, 2048};
    int m[N] = {6,    8,    16,   32};

    int devID = 0;
    hipSetDevice(devID);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, devID);
    printf("Device : %s, compute SM %d.%d.\n",devProp.name, devProp.major, devProp.minor);

	hipblasHandle_t handle;
	checkcuBlasError(hipblasCreate(&handle));

    FILE *output = NULL;
    char filename[20] = "result.txt";

	hipEvent_t start, stop;
	float time_used = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	char *d_A, *d_B; 
	int  *d_C;	// note the result is accumulated in int
	char *h_A, *h_B;
	int  *h_C;	// note the result is accumulated in int


	if ((output = fopen(filename, "w")) == NULL)
	{
		printf("Can not open file : %s\n", filename);
		exit(1);
	}
    fprintf(output, "m      \t k      \t n      \t Time      \t TFlops\n");

    for (int i=0; i<N; i++)
    {
        // allocate memory
        h_A = (char*)malloc(sizeof(char) * m[i] * k[i]);
        h_B = (char*)malloc(sizeof(char) * k[i] * n[i]);
        h_C = (int *)malloc(sizeof(int ) * m[i] * n[i]);

        checkCudaAPIErrors(hipMalloc((void **)&d_A, sizeof(char) * m[i] * k[i]));
        checkCudaAPIErrors(hipMalloc((void **)&d_B, sizeof(char) * k[i] * n[i]));
        checkCudaAPIErrors(hipMalloc((void **)&d_C, sizeof(int ) * m[i] * n[i]));

        // initilize data on host
        initArray(h_A, m[i] * k[i]);    // init the matrix to 1
        initArray(h_B, k[i] * n[i]);    // init the matrix to 1

        printf("h_A[0] = %x, h_A[last] = %x \n", h_A[0], h_A[m[i]*k[i]-1]);
        printf("h_B[0] = %x, h_B[last] = %x \n", h_B[0], h_B[k[i]*n[i]-1]);

        // copy date from host to device
        checkCudaAPIErrors(hipMemcpy(d_A, h_A, sizeof(char) * m[i] * k[i],hipMemcpyHostToDevice));
        checkCudaAPIErrors(hipMemcpy(d_B, h_B, sizeof(char) * k[i] * n[i],hipMemcpyHostToDevice));

        // gpu warm up
        cublasStat=hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n[i], m[i], k[i], 
                    &alpha, d_B, HIP_R_8I, n[i], d_A, HIP_R_8I, k[i], &beta, d_C, HIP_R_32I, n[i],
					HIP_R_32I,				// specify the computatioin type for cublasGemmEx
					HIPBLAS_GEMM_DEFAULT);		// specify the algorithm for cublasGemmEx
					//CUBLAS_GEMM_ALGO2);		// specify the algorithm for cublasGemmEx
		checkcuBlasError(cublasStat);

        hipEventRecord(start, 0);

        for (int t = 0; t < iters; t++)
        {
        cublasStat=hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n[i], m[i], k[i], 
                    &alpha, d_B, HIP_R_8I, n[i], d_A, HIP_R_8I, k[i], &beta, d_C, HIP_R_32I, n[i],
					HIP_R_32I,				// specify the computatioin type for cublasGemmEx
					HIPBLAS_GEMM_DEFAULT);		// specify the algorithm for cublasGemmEx
					//CUBLAS_GEMM_ALGO2);		// specify the algorithm for cublasGemmEx

		checkcuBlasError(cublasStat);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_used, start, stop);
        std::cout << "cublas sgemm elapsed time: " << time_used / iters << " ms" << std::endl;

        checkCudaAPIErrors(hipMemcpy(h_C, d_C, sizeof(int) * m[i] * n[i],hipMemcpyDeviceToHost));

        // verify the result computed on GPU with int8
        for (int ii=0; ii<m[i]*n[i]; ii++)
        {
            if (h_C[ii] != k[i]*Value*Value)
            {
                printf("get error result on GPU with m = %d, n = %d, C[] = %d\n", m[i], n[i], h_C[ii]);
                break;
            }
        }

        time_used /= (iters);
        TFlops     = (long(2))*m[i]*n[i]*k[i]/(time_used * 1000 * 1000 * 1000); // unit: Tflops
        fprintf(output, "%6d\t%6d\t%6d\t%10.6f\t%10.6f\n", m[i], k[i], n[i], time_used/iters, TFlops);

        checkCudaAPIErrors(hipMemcpy(h_C, d_C, sizeof(int) * m[i] * n[i],hipMemcpyDeviceToHost));
        // free memory
        free(h_A);
        free(h_B);
        free(h_C);
        checkCudaAPIErrors(hipFree(d_A));
        checkCudaAPIErrors(hipFree(d_B));
        checkCudaAPIErrors(hipFree(d_C));
    }

	hipblasDestroy(handle);
	hipEventDestroy(start);
	hipEventDestroy(stop);
    fclose(output);

	return 0;
}