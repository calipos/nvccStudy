#include "hip/hip_runtime.h"

#include "opencv/cv.hpp"
#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include <iostream>
//-I/media/hdd/lbl_trainData/softwarePackage/opencv2413/include  -I/media/hdd/lbl_trainData/softwarePackage/opencv2413/include/opencv  -L/media/hdd/lbl_trainData/softwarePackage/opencv2413/lib -lopencv_core  -lopencv_highgui  -lopencv_imgproc

#define DIM (512)
#define HALFDIM (DIM/2)
#define SOPHERE_NUM (200)
#define rnd(x) (rand()%100*0.01*x)

typedef struct sphere
{
	float r,g,b;
	float x,y,z;
	float rad,rSqr;
	__device__ float hitNearest(int ox,int oy,float*scale)
	{
		float distanceFromCenterSqr = ((ox-x)*(ox-x)+(oy-y)*(oy-y));
		if (distanceFromCenterSqr<=rSqr)
		{
			float dz=sqrt(rSqr-distanceFromCenterSqr);
			*scale=dz/rad;
			return z+dz;
		}
		else
		{
			scale=0;
			return -1;
		}
	}
	void show(){std::cout<<r<<", "<<g<<", "<<b<<", "<<x<<", "<<y<<", "<<z<<", "<<rad<<", "<<rSqr<<" "<<std::endl;}
}sphere;

__constant__ sphere dev_sphere[SOPHERE_NUM];

__global__  void rayTracing(void*img_data)
{


	int x =threadIdx.x+blockIdx.x*blockDim.x;
	int y =threadIdx.y+blockIdx.y*blockDim.y;
	int offset=x+DIM*y;
	float nearest=-1;
	unsigned char*imgData1=(unsigned char*)img_data;
	unsigned char*imgData2=(unsigned char*)img_data+DIM*DIM;
	unsigned char*imgData3=(unsigned char*)img_data+DIM*DIM*2;
	for(int i=0;i<SOPHERE_NUM;i++)
	{
		float scale_=0;
		float distance = (dev_sphere)[i].hitNearest(x-HALFDIM,y-HALFDIM,&scale_);
		if(distance>nearest)
		{
			nearest=distance;
			imgData1[offset]=scale_*(dev_sphere)[i].b;
			imgData2[offset]=scale_*(dev_sphere)[i].g;
			imgData3[offset]=scale_*(dev_sphere)[i].r;
		}
	}
}

int main()
{
	sphere*spheres=(sphere*)malloc(SOPHERE_NUM*sizeof(sphere));	
	for(int i=0;i<SOPHERE_NUM;i++)
	{
		spheres[i].x=rnd(DIM)-HALFDIM;
		spheres[i].y=rnd(DIM)-HALFDIM;
		spheres[i].z=rnd(DIM)-HALFDIM;
		spheres[i].r=rnd(255);
		spheres[i].g=rnd(255);
		spheres[i].b=rnd(255);
		spheres[i].rad=rnd(100)+20;
		spheres[i].rSqr=spheres[i].rad*spheres[i].rad;
		//spheres[i].show();
	}

	hipMemcpyToSymbol(HIP_SYMBOL(dev_sphere),spheres,SOPHERE_NUM*sizeof(sphere));
	free(spheres);
	
	
	cv::Mat img = cv::Mat::zeros(DIM,DIM,CV_8UC3);
	unsigned char *dev_img=NULL;
	hipMalloc((void**)&dev_img,3*DIM*DIM*sizeof(unsigned char));
	dim3 gridShape(DIM/16,DIM/16);
	dim3 blockShape(16,16);
	rayTracing<<<gridShape,blockShape>>>((void*)dev_img);
	
	
	//hipMemcpy(img.data,dev_img,3*DIM*DIM*sizeof(unsigned char),hipMemcpyDeviceToHost);

	unsigned char* tmp=(unsigned char*)malloc(3*DIM*DIM*sizeof(unsigned char));
	hipMemcpy(tmp,dev_img,3*DIM*DIM*sizeof(unsigned char),hipMemcpyDeviceToHost);
	int dim1_=DIM*DIM;
	int dim2_=2*dim1_;
	int pixel=0;
	for(int i=0;i<img.rows;i++)
	{
		for(int j=0;j<img.cols;j++)
		{
			img.at<cv::Vec3b>(i,j)[0]=tmp[pixel];
			img.at<cv::Vec3b>(i,j)[1]=tmp[pixel+dim1_];
			img.at<cv::Vec3b>(i,j)[2]=tmp[pixel+dim2_];
			pixel++;
		}
	}

	hipFree(dev_sphere);
	hipFree(dev_img);
	cv::imwrite("1.jpg",img);
	cv::imshow("123",img);
	cv::waitKey();
	return 0;
}