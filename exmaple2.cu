
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void add(int a,int b, int*c)
{
    *c=a+b;
}

int main()
{
    int c=0;
    int *dev_c=NULL;
    std::cout<<"hipMalloc ret = "<<hipMalloc((void**)&dev_c,sizeof(int))<<std::endl;
	
	add<<<1,1>>>(2,7,dev_c);
	
    std::cout<<"hipMemcpy ret = "<<hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost)<<std::endl;
	std::cout<<c<<std::endl;
	std::cout<<"hipFree ret = "<<hipFree(dev_c)<<std::endl;
    std::cout<<123<<std::endl;
    return 0;
}