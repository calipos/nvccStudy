
#include <hip/hip_runtime.h>
#include <iostream>



int main()
{
	int count=0;
	std::cout<<"hipGetDeviceCount ret = "<<hipGetDeviceCount(&count)<<std::endl;
	std::cout<<"device count = "<<count<<std::endl;
	hipDeviceProp_t prop;
	for(int i=0;i<count;i++)
	{
		
		std::cout<<"hipGetDeviceProperties ret = "<<hipGetDeviceProperties(&prop,i)<<std::endl;
		
		std::cout<<"---general device info for device "<<i<<"---"<<std::endl;
		std::cout<<"Name : "<<prop.name<<std::endl;
		std::cout<<"compute capability : "<<prop.major<<" "<<prop.minor<<std::endl;
		
		
	}
	
    std::cout<<123<<std::endl;
    return 0;
}